#include "hip/hip_runtime.h"
/* A test program of solving the 3D Poisson equation 
       -div (grad u) = f
   using the 4th-order accurate 19-point cell. The solution domain
   is that unit cube. The right-hand side function is
       f(x,y,z) = 3*pi^2*sin(pi*x)*sin(pi*y)*sin(pi*z),
   so that the analytical solution should be
       u(x,y,z)=sin(pi*x)*sin(pi*y)*sin(pi*z)
   Jacobi iterations are used as the numerical solution method.
*/
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <math.h>
//#include <omp.h>
#include <sys/time.h>
#include <assert.h>
#define REAL double
#define FLOPS 14.0
#define chunk 64 
const double kMicro = 1.0e-6;

double ***alloc3D(int n,int m,int k)
{
  double ***m_buffer = ((void *)0);
  int nx = n;
  int ny = m;
  int nk = k;
  m_buffer = ((double ***)(malloc(sizeof(double **) * nk)));
  m_buffer?((void )0) : ((__assert_fail("m_buffer","poisson19.c",33,__PRETTY_FUNCTION__) , ((void )0)));
  double **m_tempzy = (double **)(malloc(sizeof(double *) * nk * ny));
  double *m_tempzyx = (double *)(malloc(sizeof(double ) * nx * ny * nk));
  int z;
  int y;
  for (z = 0; z < nk; (z++ , m_tempzy += ny)) {
    m_buffer[z] = m_tempzy;
    for (y = 0; y < ny; (y++ , m_tempzyx += nx)) {
      m_buffer[z][y] = m_tempzyx;
    }
  }
  return m_buffer;
}

void free3D(double ***E)
{
  free(E[0][0]);
  free(E[0]);
  free(E);
}

double getTime()
{
  struct timeval TV;
  const int RC = gettimeofday(&TV,((void *)0));
  if (RC == -1) {
    printf("ERROR: Bad call to gettimeofday\n");
    return (-1);
  }
  return ((double )TV . tv_sec) + kMicro * ((double )TV . tv_usec);
// end getTime()                                                                               
}
static void mint_1_1527(int n,hipPitchedPtr dev_1_u_old,hipPitchedPtr dev_2_u_new,hipPitchedPtr dev_3_rhs,double factor,double factor2,int num2blockDim_1_1527,float invYnumblockDim_1_1527);

int main(int nargs,char **args)
{
/* number of points in each direction */
  int n;
/* grid spacing, same in all the directions */
  double h;
  double ***u_old;
  double ***u_new;
  double ***rhs;
  double factor;
  double factor2;
  double l2_norm;
  int i;
  int j;
  int k;
  int max_iters = 100;
  if (nargs > 1) {
    n = atoi(args[1]);
  }
  else {
    n = 256;
  }
  h = 1.0 / (n - 1);
  u_old = alloc3D(n + 2,n + 2,n + 2);
  u_new = alloc3D(n + 2,n + 2,n + 2);
  rhs = alloc3D(n + 2,n + 2,n + 2);
/* fill the right-hand side vector */
/* use deferred correction */
  factor = (1.0 - h * h * 3.14159265358979323846 * 3.14159265358979323846 / 4) * 3.0 * 3.14159265358979323846 * 3.14159265358979323846;
  for (k = 0; k <= n + 1; k++) 
    for (j = 0; j <= n + 1; j++) 
      for (i = 0; i <= n + 1; i++) 
        rhs[k][j][i] = 6. * h * h * factor * sin(3.14159265358979323846 * i * h) * sin(3.14159265358979323846 * j * h) * sin(3.14159265358979323846 * k * h);
/* use initial zero guess */
  for (k = 0; k <= n + 1; k++) 
    for (j = 0; j <= n + 1; j++) 
      for (i = 0; i <= n + 1; i++) 
        u_old[k][j][i] = u_new[k][j][i] = 0.;
/* Jacobi iterations */
  l2_norm = 1e+12;
  factor = 1.0 / 24;
  factor2 = 6. * h * h;
  printf("\n=====Timings (sec) for 19-Point Jacobi, Solving Poisson Eqn ");
  if (sizeof(double ) == 4) {
    printf(" (Single Precision) =====\n");
  }
  if (sizeof(double ) == 8) {
    printf(" (Double Precision) =====\n");
  }
  printf("Kernel\t Time(sec)\tGflops  \tBW-ideal(GB/s)\tBW-algorithm (N=(%d) iters=%d)\n",n,max_iters);
  printf("------\t----------\t--------\t--------------\t------------\n");
  int nIters = 0;
  double time_elapsed = getTime();
  double Gflops = 0.0;
/* Mint: Replaced Pragma: #pragma mint copy( u_old, toDevice,( n+2 ), n+2,( n+2 ) ) */
  hipError_t stat_dev_1_u_old;
  hipExtent ext_dev_1_u_old = make_hipExtent(((n+2)) * sizeof(double ),(n+2),((n+2))));
/* Mint: Malloc on the device */
  hipPitchedPtr dev_1_u_old;
  stat_dev_1_u_old = hipMalloc3D(&dev_1_u_old,ext_dev_1_u_old);
  if (stat_dev_1_u_old != hipSuccess) 
    fprintf(stderr,"%s\n",hipGetErrorString(stat_dev_1_u_old));
/* Mint: Copy host to device */
  hipMemcpy3DParms param_1_dev_1_u_old = {0};
  param_1_dev_1_u_old . srcPtr = make_hipPitchedPtr(((void *)u_old[0][0]),((n+2)) * sizeof(double ),((n+2)),(n+2));
  param_1_dev_1_u_old . dstPtr = dev_1_u_old;
  param_1_dev_1_u_old . extent = ext_dev_1_u_old;
  param_1_dev_1_u_old . kind = hipMemcpyHostToDevice;
  stat_dev_1_u_old = hipMemcpy3D(&param_1_dev_1_u_old);
  if (stat_dev_1_u_old != hipSuccess) 
    fprintf(stderr,"%s\n",hipGetErrorString(stat_dev_1_u_old));
/* Mint: Replaced Pragma: #pragma mint copy( u_new, toDevice,( n+2 ), n+2,( n+2 ) ) */
  hipError_t stat_dev_2_u_new;
  hipExtent ext_dev_2_u_new = make_hipExtent(((n+2)) * sizeof(double ),(n+2),((n+2))));
/* Mint: Malloc on the device */
  hipPitchedPtr dev_2_u_new;
  stat_dev_2_u_new = hipMalloc3D(&dev_2_u_new,ext_dev_2_u_new);
  if (stat_dev_2_u_new != hipSuccess) 
    fprintf(stderr,"%s\n",hipGetErrorString(stat_dev_2_u_new));
/* Mint: Copy host to device */
  hipMemcpy3DParms param_2_dev_2_u_new = {0};
  param_2_dev_2_u_new . srcPtr = make_hipPitchedPtr(((void *)u_new[0][0]),((n+2)) * sizeof(double ),((n+2)),(n+2));
  param_2_dev_2_u_new . dstPtr = dev_2_u_new;
  param_2_dev_2_u_new . extent = ext_dev_2_u_new;
  param_2_dev_2_u_new . kind = hipMemcpyHostToDevice;
  stat_dev_2_u_new = hipMemcpy3D(&param_2_dev_2_u_new);
  if (stat_dev_2_u_new != hipSuccess) 
    fprintf(stderr,"%s\n",hipGetErrorString(stat_dev_2_u_new));
/* Mint: Replaced Pragma: #pragma mint copy( rhs, toDevice,( n+2 ), n+2,( n+2 ) ) */
  hipError_t stat_dev_3_rhs;
  hipExtent ext_dev_3_rhs = make_hipExtent(((n+2)) * sizeof(double ),(n+2),((n+2))));
/* Mint: Malloc on the device */
  hipPitchedPtr dev_3_rhs;
  stat_dev_3_rhs = hipMalloc3D(&dev_3_rhs,ext_dev_3_rhs);
  if (stat_dev_3_rhs != hipSuccess) 
    fprintf(stderr,"%s\n",hipGetErrorString(stat_dev_3_rhs));
/* Mint: Copy host to device */
  hipMemcpy3DParms param_3_dev_3_rhs = {0};
  param_3_dev_3_rhs . srcPtr = make_hipPitchedPtr(((void *)rhs[0][0]),((n+2)) * sizeof(double ),((n+2)),(n+2));
  param_3_dev_3_rhs . dstPtr = dev_3_rhs;
  param_3_dev_3_rhs . extent = ext_dev_3_rhs;
  param_3_dev_3_rhs . kind = hipMemcpyHostToDevice;
  stat_dev_3_rhs = hipMemcpy3D(&param_3_dev_3_rhs);
  if (stat_dev_3_rhs != hipSuccess) 
    fprintf(stderr,"%s\n",hipGetErrorString(stat_dev_3_rhs));
{
    int iters = 0;
    while(iters < max_iters && l2_norm > 1e-9){
      ++iters;
/* update each interior point */
      
#pragma mint for nest(all) tile(16,16,1)
      int num3blockDim_1_1527 = (n - 1 + 1) % 1 == 0?(n - 1 + 1) / 1 : (n - 1 + 1) / 1 + 1;
      int num2blockDim_1_1527 = (n - 1 + 1) % 16 == 0?(n - 1 + 1) / 16 : (n - 1 + 1) / 16 + 1;
      int num1blockDim_1_1527 = (n - 1 + 1) % 16 == 0?(n - 1 + 1) / 16 : (n - 1 + 1) / 16 + 1;
      float invYnumblockDim_1_1527 = 1.00000F / num2blockDim_1_1527;
      dim3 blockDim_1_1527(16,16,1);
      dim3 gridDim_1_1527(num1blockDim_1_1527,num2blockDim_1_1527*num3blockDim_1_1527);
      mint_1_1527<<<gridDim_1_1527,blockDim_1_1527>>>(n,dev_1_u_old,dev_2_u_new,dev_3_rhs,factor,factor2,num2blockDim_1_1527,invYnumblockDim_1_1527);
      hipDeviceSynchronize();
      hipError_t err_mint_1_1527 = hipGetLastError();
      if (err_mint_1_1527) {
        fprintf(stderr,"In %s, %s\n","mint_1_1527",hipGetErrorString(err_mint_1_1527));
      }
/* pointer swap */
      
#pragma mint single
{
        double ***tmp;
        void *dev_tmp;
        dev_tmp = dev_1_u_old . ptr;
        dev_1_u_old . ptr = dev_2_u_new . ptr;
        dev_2_u_new . ptr = dev_tmp;
        nIters = iters;
      }
    }
  }
/* Mint: Replaced Pragma: #pragma mint copy( u_old, fromDevice,( n+2 ),( n+2 ),( n+2 ) ) */
/* Mint: Copy device to host */
  hipMemcpy3DParms param_4_dev_1_u_old = {0};
  param_4_dev_1_u_old . srcPtr = dev_1_u_old;
  param_4_dev_1_u_old . dstPtr = make_hipPitchedPtr(((void *)u_old[0][0]),((n+2)) * sizeof(double ),((n+2)),((n+2)));
  param_4_dev_1_u_old . extent = ext_dev_1_u_old;
  param_4_dev_1_u_old . kind = hipMemcpyDeviceToHost;
  stat_dev_1_u_old = hipMemcpy3D(&param_4_dev_1_u_old);
  if (stat_dev_1_u_old != hipSuccess) 
    fprintf(stderr,"%s\n",hipGetErrorString(stat_dev_1_u_old));
  hipFree(dev_1_u_old . ptr);
  hipFree(dev_2_u_new . ptr);
  hipFree(dev_3_rhs . ptr);
  time_elapsed = getTime() - time_elapsed;
  Gflops = ((double )((nIters * n * n * n) * 1e-9 * 14.0)) / time_elapsed;
  l2_norm = 0;
  for (k = 0; k <= n + 1; k++) 
    for (j = 0; j <= n + 1; j++) 
      for (i = 0; i <= n + 1; i++) {
        factor = sin(3.14159265358979323846 * i * h) * sin(3.14159265358979323846 * j * h) * sin(3.14159265358979323846 * k * h);
        l2_norm += (factor - u_old[k][j][i]) * (factor - u_old[k][j][i]);
      }
  printf("%s%3.3f \t%5.3f\n","Poisson19   ",time_elapsed,Gflops);
  printf(":N %d M %d K %d , iteration %d\n",n,n,n,nIters);
  printf(":max: %20.12e, l2norm: %20.12e\n",factor,sqrt(l2_norm * h * h * h));
//printf("Total iterations used: %d, l2-norm of error=%e\n",
//	 nIters,sqrt(l2_norm*h*h*h));
  free3D(u_new);
  free3D(u_old);
  free3D(rhs);
  return 0;
}

__global__ static void mint_1_1527(int n,hipPitchedPtr dev_1_u_old,hipPitchedPtr dev_2_u_new,hipPitchedPtr dev_3_rhs,double factor,double factor2,int num2blockDim_1_1527,float invYnumblockDim_1_1527)
{
#define TILE_X 16
#define TILE_Y 16
  __device__ __shared__ double _sh_block_u_old[3][TILE_Y + 2][TILE_X + 2];
  double *u_old = (double *)dev_1_u_old . ptr;
  int _width = dev_1_u_old . pitch / sizeof(double );
  int _slice = dev_1_u_old . ysize * _width;
  double *u_new = (double *)dev_2_u_new . ptr;
  double *rhs = (double *)dev_3_rhs . ptr;
  float blocksInY = num2blockDim_1_1527;
  float invBlocksInY = invYnumblockDim_1_1527;
  int _p_i;
  int _p_j;
  int _p_k;
{
    int _upperb_y = n;
    int _upperb_x = n;
    int _idx = threadIdx.x + 1;
    int _gidx = _idx + blockDim.x * blockIdx.x;
    int _idy = threadIdx.y + 1;
    int _gidy = _idy + blockDim.y * 1 * blockIdx.y;
    int _idz = threadIdx.z + 1;
    int blockIdxz = blockIdx.y * invBlocksInY;
    int blockIdxy = blockIdx.y - blockIdxz * blocksInY;
    _gidy = _idy + blockIdxy * blockDim.y;
    int _gidz = _idz + blockIdxz * blockDim.z;
    int _index3D = _gidx + _gidy * _width + _gidz * _slice;
    _idz = 1;
    _idy = threadIdx.y + 1;
    _idx = threadIdx.x + 1;
    int _borderIdx = _idx;
    int _borderIdy = 0;
    int _borderGlobalIndexDiff = 0;
    _borderIdx = (threadIdx.y == 1?0 : _borderIdx);
    _borderIdx = (threadIdx.y == 2?blockDim.x + 1 : _borderIdx);
    _borderIdy = (threadIdx.y == 3?blockDim.y + 1 : _borderIdy);
    _borderIdy = (threadIdx.y == 1 || threadIdx.y == 2?_idx : _borderIdy);
    _borderGlobalIndexDiff = _borderIdx - _idx + _width * (_borderIdy - _idy);
    int _borderGlobalIndexDiff_up = _borderGlobalIndexDiff - _slice;
    int _borderGlobalIndexDiff_down = _borderGlobalIndexDiff + _slice;
    if (threadIdx.y < 4 * 1) 
      _sh_block_u_old[_idz - 1][_borderIdy][_borderIdx] = u_old[_index3D + _borderGlobalIndexDiff_up];
    if (threadIdx.y < 4 * 1) 
      _sh_block_u_old[_idz][_borderIdy][_borderIdx] = u_old[_index3D + _borderGlobalIndexDiff];
{
      if (_gidz >= 1 && _gidz <= n) {{{
            if (_gidy >= 1 && _gidy <= n) {{{
                  if (_gidx >= 1 && _gidx <= n) {{
                      _sh_block_u_old[_idz + -1][_idy][_idx] = u_old[_index3D - _slice];
                      _sh_block_u_old[_idz + 1][_idy][_idx] = u_old[_index3D + _slice];
                      if (threadIdx.x == 0 && threadIdx.y == 0) {
                        u_old = u_old + _index3D - (1 + _width);
                        _sh_block_u_old[_idz][0][0] = u_old[0];
                        _sh_block_u_old[_idz][0][blockDim.x + 1] = u_old[blockDim.x + 1];
                        _sh_block_u_old[_idz][blockDim.y + 1][0] = u_old[_width * (blockDim.y + 1)];
                        _sh_block_u_old[_idz][blockDim.y + 1][blockDim.x + 1] = u_old[_width * (blockDim.y + 1) + (blockDim.x + 1)];
                        u_old = u_old - _index3D + (1 + _width);
                      }
                      if (threadIdx.y < 4 * 1) 
                        _sh_block_u_old[_idz + 1][_borderIdy][_borderIdx] = u_old[_index3D + _borderGlobalIndexDiff_down];
                      double _ru_old = u_old[_index3D];
                      _sh_block_u_old[_idz + 0][_idy][_idx] = _ru_old;
                      double _rrhs = rhs[_index3D];
                      double _ru_new;
                      __syncthreads();
                      _ru_new = factor * (_rrhs + factor2 * (_sh_block_u_old[_idz][_idy][_idx - 1] + _sh_block_u_old[_idz][_idy][_idx + 1] + _sh_block_u_old[_idz][_idy - 1][_idx] + _sh_block_u_old[_idz][_idy + 1][_idx] + _sh_block_u_old[_idz + 1][_idy][_idx] + _sh_block_u_old[_idz - 1][_idy][_idx]) + _sh_block_u_old[_idz - 1][_idy - 1][_idx] + _sh_block_u_old[_idz - 1][_idy + 1][_idx] + _sh_block_u_old[_idz - 1][_idy][_idx - 1] + _sh_block_u_old[_idz - 1][_idy][_idx + 1] + _sh_block_u_old[_idz][_idy - 1][_idx - 1] + _sh_block_u_old[_idz][_idy + 1][_idx - 1] + _sh_block_u_old[_idz][_idy - 1][_idx + 1] + _sh_block_u_old[_idz][_idy + 1][_idx + 1] + _sh_block_u_old[_idz + 1][_idy - 1][_idx] + _sh_block_u_old[_idz + 1][_idy + 1][_idx] + _sh_block_u_old[_idz + 1][_idy][_idx - 1] + _sh_block_u_old[_idz + 1][_idy][_idx + 1]);
                      u_new[_index3D] = _ru_new;
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
  }
}
